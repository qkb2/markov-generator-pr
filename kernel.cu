﻿#include "Generator.h"
#include "MarkovGenerator.h"
#include "MarkovGeneratorMP.h"
#include "MarkovGeneratorCUDA.cuh"

int main(int argc, char* argv[]) {
	if (argc < 3) {
		return -1;
	}
	std::string path = argv[1];
	std::string opt = argv[2];
	std::string starter = "a backdrop";
	Generator * generator;
	if (opt.compare("CUDA") == 0) {
		generator = new MarkovGeneratorCUDA(3, path);
	}
	else if (opt.compare("MP") == 0) {
		generator = new MarkovGeneratorMP(3, path);
	}
	else {
		generator = new MarkovGenerator(3, path);
	}
	generator->generate_ngram_markov();
	auto text = generator->generate_word_chain(starter, 100);
	std::cout << text;
}