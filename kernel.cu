﻿#include "MarkovGenerator.h"

int main(int argc, char* argv[]) {
	std::string path = argv[1];
	std::string starter = "a backdrop";
	auto generator = new MarkovGenerator(3, path);
	generator->generate_ngram_markov();
	auto text = generator->generate_word_chain(starter, 100);
	std::cout << text;
}